#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

__global__ void d_matrix_multi(float * d_matrix_a_ptr_in, float * d_matrix_b_ptr_in, float * d_matrix_c_ptr_in, int d_row_size_a_in, int d_row_size_b_in, int d_column_size_c_in)
{
	int d_i=blockIdx.x*blockDim.x+threadIdx.x;
	int d_j=blockIdx.y*blockDim.y+threadIdx.y;
	int d_k;
	float d_sum=(float)0.0;

	for (d_k=0;d_k<d_row_size_b_in;d_k++)
	{
		d_sum+=d_matrix_a_ptr_in[d_i*d_row_size_b_in+d_k]*d_matrix_b_ptr_in[d_k*d_column_size_c_in+d_j];		
	}
		d_matrix_c_ptr_in[d_i*d_column_size_c_in+d_j]=d_sum;
}

int h_matrix_init(float * h_matrix_ptr_in, int h_row_size_in, int h_column_size_in)
{
	int h_i, h_j;

	if(h_matrix_ptr_in==NULL)
	{
		fprintf(stderr, "INVALID MATRIX POINTER.\n");
		return 1;
	}
	else
	{
		srand((unsigned)time(NULL)); 
		for(h_i=0; h_i<h_row_size_in;h_i++)
		{
			for(h_j=0;h_j<h_column_size_in;h_j++)
			{
				h_matrix_ptr_in[h_i*h_column_size_in+h_j]=((float) rand()/(float) RAND_MAX);
			}
		}
		return 0;
	}

}

int h_display_result(float * h_matrix_ptr_in, int h_row_size_in, int h_column_size_in)
{
	int h_i, h_j;

	if (h_matrix_ptr_in==NULL)
	{
		fprintf(stderr,"ERROR IN MATRIX POINTER INPUT.\n");
	   	return 1;
	}
	else if(h_row_size_in==0 || h_column_size_in==0)	
	{
		fprintf(stderr, "ERROR IN MATRIX SIZE INPUT.\n");
		return 1;
	}
	else
	{
		for(h_i=0;h_i<h_row_size_in;h_i++)
		{
			for(h_j=0;h_j<h_column_size_in;h_j++)
			{
				fprintf(stdout,"C[%d][%d]=%f.\n",h_i,h_j,h_matrix_ptr_in[h_i*h_column_size_in+h_j]);
			}
		}

		return 0;
	}
}

int main(int argc, char **argv)
{
	int h_row_size_a, h_row_size_b, h_row_size_c;
	int h_column_size_a, h_column_size_b, h_column_size_c;

	float * h_matrix_a_ptr;
	float * h_matrix_b_ptr;
	float * h_matrix_c_ptr;

	int h_ret=0;

	if(argc!=5)
	{
		fprintf(stderr, "ERROR IN USAGE.\n");
		fprintf(stderr,"./matrix row_size_a column_size_a row_size_b column_size_b \n");
		return 1;
	}
	else
	{	
		h_row_size_a=atoi(argv[1]);
		h_column_size_a=atoi(argv[2]);
		h_row_size_b=atoi(argv[3]);
		h_column_size_b=atoi(argv[4]);

		if((h_row_size_a==0 || h_column_size_a==0 || h_row_size_b==0 || h_column_size_b==0) || (h_column_size_a!=h_row_size_b))
		{
			fprintf(stderr, "INVAILD MATRIX SIZE.\n");
			fprintf(stderr, "C=AxB.\n");
			fprintf(stderr, "Dim for Matrix A is %d x %d.\n", h_row_size_a,h_column_size_a);
			fprintf(stderr, "Dim for Matrix b is %d x %d.\n", h_row_size_b,h_column_size_b);
			return 1;
		}
		else
		{	
			//MATRIX SIZE C
			h_row_size_c=h_row_size_a;
			h_column_size_c=h_column_size_b;

			//UNIFIED MEMORY
			hipMallocManaged(&h_matrix_a_ptr,h_row_size_a*h_column_size_a*sizeof(float));			}
			hipMallocManaged(&h_matrix_b_ptr,h_row_size_b*h_column_size_b*sizeof(float));
			hipMallocManaged(&h_matrix_c_ptr,h_row_size_c*h_column_size_c*sizeof(float));

			//HOST MATRIX INITIALIZATION
			h_ret=h_matrix_init((float *)h_matrix_a_ptr,h_row_size_a, h_column_size_a);
			if(h_ret!=0)
			{
				fprintf(stderr, "MATRIX A INITIALIZATION ERROR.\n");
				return 1;
			}
			h_ret=h_matrix_init((float *)h_matrix_b_ptr, h_row_size_b,h_column_size_b);
			if(h_ret!=0)
			{
				fprintf(stderr, "MATRIX B INITIALIZATION ERROR.\n");
				return 1;
			}
			
		}

		
		//CUDA KERNEL CALL
		dim3 threadsPerBlock(16,16);
		dim3 numBlocks(h_row_size_c/16, h_column_size_c/16);
		d_matrix_multi<<<numBlocks, threadsPerBlock>>>(h_matrix_a_ptr,h_matrix_b_ptr, h_matrix_c_ptr, h_row_size_a, h_column_size_b, h_column_size_c);
		
		//DEVICE SYNCHRONIZATION
		hipDeviceSynchronize();
		
		//DISPLAY RESULT
		h_ret=h_display_result((float *)h_matrix_c_ptr,h_row_size_c, h_column_size_c);

		//free
		hipFree(h_matrix_a_ptr);
		hipFree(h_matrix_b_ptr);
		hipFree(h_matrix_c_ptr);
	}
	return 0;
}
